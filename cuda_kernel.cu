#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <hip/hip_math_constants.h>
#include "cuda_kernel.h"

#define FLOAT_DT float
#define coor_2d(COOR1, COOR2, MAX_COOR1, MAX_COOR2) COOR1 + COOR2 * MAX_COOR1
#define coor_3d(COOR1, COOR2, COOR3, MAX_COOR1, MAX_COOR2, MAX_COOR3) COOR1 + COOR2 * MAX_COOR1 + COOR3 * MAX_COOR1 * MAX_COOR2
#define MAX(a, b) ((a > b) ? (a) : (b))

__global__ void calculate_pixel(const FLOAT_DT *delay1, const FLOAT_DT *delay2, const FLOAT_DT *wavel1, const FLOAT_DT *wavel2,
                     const FLOAT_DT *image_real, const FLOAT_DT *image_imag, FLOAT_DT *output, int max_x, int max_y) {
    // Coordinates
    long long threadID = threadIdx.x+blockIdx.x*blockDim.x;
    int x = threadID % max_y;
    int y = (threadID/max_y)%max_x;
    int z = threadID/(max_x*max_y);

    // Get value of ps1
    FLOAT_DT value = HIP_PI_F * 2.0 * delay1[z] / wavel1[x];
    if (max_y % 2 != 1 && max_y / 2 == z && x == 0) {
        value = 0;
    }
    FLOAT_DT ps1_value_real, ps1_value_imaginary;
    sincosf(value, &ps1_value_imaginary, &ps1_value_real);

    // Get value of ps2
    value = HIP_PI_F * 2.0 * delay2[z] / wavel2[y];
    if (max_x % 2 != 1 && max_x / 2 == z && y == 0) {
        value = 0;
    }
    FLOAT_DT ps2_value_real, ps2_value_imaginary;
    sincosf(value, &ps2_value_imaginary, &ps2_value_real);

    // Final multiplication
    FLOAT_DT total_real = ps1_value_real * ps2_value_real + ps1_value_imaginary * ps2_value_imaginary;
    FLOAT_DT total_imaginary = ps1_value_real * ps2_value_imaginary - ps1_value_imaginary * ps2_value_real;

    atomicAdd(&output[z], (image_real[x + y * max_y] * total_real - image_imag[x+y*max_y]*total_imaginary) * sqrtf(total_real * total_real + total_imaginary * total_imaginary));
}

// image: 2d, delay1_in: 1d, delay2_in: 1d
// delay1 und delay2 werden implizit transponiert
void AmplitudeExtractionFourier2D(FLOAT_DT *image_real, FLOAT_DT *image_imag, FLOAT_DT *delay1, FLOAT_DT *delay2, FLOAT_DT fs_in, FLOAT_DT *output,
                                  int max_x, int max_y) {
    int longest_dimension;
    FLOAT_DT fs;

    longest_dimension = MAX(max_x, max_y);
    fs = fs_in - fs_in / (FLOAT_DT) longest_dimension;

    FLOAT_DT *wavel1, *wavel2;
    wavel1 = (FLOAT_DT*)malloc(max_y*sizeof(FLOAT_DT));
    wavel2 = (FLOAT_DT*)malloc(max_x*sizeof(FLOAT_DT));

    // Create wavel1
    if (max_y % 2 == 1) {
        for (int i = 0; i < max_y; i++) {
            // circshift + fftshift
            int shifted_i = (i + ((int) max_y / 2)) % max_y;
            wavel1[i] = 1 / ((FLOAT_DT) shifted_i * (fs / (FLOAT_DT) (max_y - 1)) - fs / 2);
        }
    } else {
        for (int i = 0; i < max_y; i++) {
            // Shift (f1(end:-1:size(CE,1)/2+2)=-f1(2:size(CE,1)/2);)
            int backwards_i = max_y - i - 1;

            if (backwards_i < max_y / 2 - 1) {
                wavel1[i] = -wavel1[backwards_i + 1];
            } else {
                wavel1[i] = 1 / ((FLOAT_DT) i * (fs / (FLOAT_DT) (max_y - 1)));
            }
        }
    }

    // Create wavel2
    if (max_x % 2 == 1) {
        for (int i = 0; i < max_x; i++) {
            // circshift + fftshift
            int shifted_i = (i + ((int) max_x / 2)) % max_x;
            wavel2[i] = 1 / ((FLOAT_DT) shifted_i * (fs / (FLOAT_DT) (max_x - 1)) - fs / 2);
        }
    } else {
        for (int i = 0; i < max_x; i++) {
            // Shift (f2(end:-1:size(CE,2)/2+2)=-f2(2:size(CE,2)/2);)
            int backwards_i = max_x - i - 1;

            if (backwards_i < max_x / 2 - 1) {
                wavel2[i] = -wavel2[backwards_i + 1];
            } else {
                wavel2[i] = 1 / ((FLOAT_DT) i * (fs / (FLOAT_DT) (max_x - 1)));
            }
        }
    }

    for (int i = 0; i < max_x * max_y; i++) {
        output[i] = 0.0f;
    }

    // Allocate space on GPU
    FLOAT_DT *gpu_delay1, *gpu_delay2, *gpu_wavel1, *gpu_wavel2, *gpu_image_real, *gpu_image_imag, *gpu_output;
    hipMalloc(&gpu_delay1, max_x*max_y*sizeof(FLOAT_DT));
    hipMalloc(&gpu_delay2, max_x*max_y*sizeof(FLOAT_DT));
    hipMalloc(&gpu_wavel1, max_y*sizeof(FLOAT_DT));
    hipMalloc(&gpu_wavel2, max_x*sizeof(FLOAT_DT));
    hipMalloc(&gpu_image_real, max_x*max_y*sizeof(FLOAT_DT));
    hipMalloc(&gpu_image_imag, max_x*max_y*sizeof(FLOAT_DT));
    hipMalloc(&gpu_output, max_x*max_y*sizeof(FLOAT_DT));

    // Copy data to GPU
    hipMemcpy(gpu_delay1, delay1, max_x*max_y*sizeof(FLOAT_DT), hipMemcpyHostToDevice);
    hipMemcpy(gpu_delay2, delay2, max_x*max_y*sizeof(FLOAT_DT), hipMemcpyHostToDevice);
    hipMemcpy(gpu_wavel1, wavel1, max_y*sizeof(FLOAT_DT), hipMemcpyHostToDevice);
    hipMemcpy(gpu_wavel2, wavel2, max_x*sizeof(FLOAT_DT), hipMemcpyHostToDevice);
    hipMemcpy(gpu_image_real, image_real, max_x*max_y*sizeof(FLOAT_DT), hipMemcpyHostToDevice);
    hipMemcpy(gpu_image_imag, image_imag, max_x*max_y*sizeof(FLOAT_DT), hipMemcpyHostToDevice);

    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
        printf("Sync kernel error1: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error1: %s\n", hipGetErrorString(errAsync));

    // 1024: max -> 1024
    int count_threads = 1024;
    long long needed_blocks = ((long long) max_x*max_y*max_x*max_y/count_threads)+1;
    calculate_pixel<<<needed_blocks, count_threads>>>(gpu_delay1,gpu_delay2,gpu_wavel1,gpu_wavel2,gpu_image_real, gpu_image_imag, gpu_output,max_x,max_y);


    errSync  = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
        printf("Sync kernel error2: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error2: %s\n", hipGetErrorString(errAsync));

    // Copy result from GPU
    hipMemcpy(output, gpu_output, max_x*max_y*sizeof(FLOAT_DT), hipMemcpyDeviceToHost);

    errSync  = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
        printf("Sync kernel error3: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error3: %s\n", hipGetErrorString(errAsync));

    hipFree(gpu_delay1);
    hipFree(gpu_delay2);
    hipFree(gpu_wavel1);
    hipFree(gpu_wavel2);
    hipFree(gpu_image_real);
    hipFree(gpu_image_imag);
    hipFree(gpu_output);
    free(wavel1);
    free(wavel2);
}
